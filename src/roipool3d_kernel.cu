/* 
Point cloud feature pooling 
Written by Yuqi Huo
All Rights Reserved 2019. 
*/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
// #define DEBUG

__device__ inline int pt_in_box3d(float x, float y, float z, float cx, float bottom_y, float cz, float h, float w,
                              float l, float angle, float max_dis){
    float x_rot, z_rot, cosa, sina, cy;
    int in_flag;
    cy = bottom_y - h / 2.0;
    if ((fabsf(x - cx) > max_dis) || (fabsf(y - cy) > h / 2.0) || (fabsf(z - cz) > max_dis)){
        return 0;
    }
    cosa = cos(angle); sina = sin(angle);
    x_rot = (x - cx) * cosa + (z - cz) * (-sina);
    z_rot = (x - cx) * sina + (z - cz) * cosa;

    in_flag = (x_rot >= -l / 2.0) & (x_rot <= l / 2.0) & (z_rot >= -w / 2.0) & (z_rot <= w / 2.0);
    return in_flag;
}


__global__ void roipool3d_forward(int batch_size, int pts_num, int boxes_num, int sampled_pts_num, 
                                  const float *xyz, const float *boxes3d, 
                                  float *pooled_features, int *pooled_empty_flag){
    // params xyz: (B, N, 3)
    // params boxes3d: (B, M, 7)
    // params pooled_features: (B, M, 512, 3+C)
    // params pooled_empty_flag: (B, M)

    int boxes_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boxes_idx >= boxes_num){
        return;
    }
    
    for (int i = 0; i < batch_size; i++){
        int cnt = 0;
        for (int k = 0; k < pts_num; k++){
            int pt_offset = i * pts_num * 3 + k * 3;
            int box_offset = i * boxes_num * 7 + boxes_idx * 7;

            int cur_in_flag = pt_in_box3d(xyz[pt_offset], xyz[pt_offset + 1], xyz[pt_offset + 2], boxes3d[box_offset], 
                                          boxes3d[box_offset + 1], boxes3d[box_offset + 2], boxes3d[box_offset + 3], 
                                          boxes3d[box_offset + 4], boxes3d[box_offset + 5], boxes3d[box_offset + 6], 10.0);
            if (cur_in_flag){
                if (cnt < sampled_pts_num){
                    int feature_out_offset = i * boxes_num * sampled_pts_num * (3) + 
                                             boxes_idx * sampled_pts_num * (3) + 
                                             cnt * (3);

                    // copy xyz
                    for (int j = 0; j < 3; j++)
                        pooled_features[feature_out_offset + j] = xyz[pt_offset + j];


                    cnt++;
                }
                else break;
            }
        }

        if (cnt == 0){
            pooled_empty_flag[i * boxes_num + boxes_idx] = 1;
        }
        else if (cnt < sampled_pts_num){
            // duplicate same points for sampling
            for (int k = cnt; k < sampled_pts_num; k++){
                int duplicate_idx = k % cnt;
                int src_offset = i * boxes_num * sampled_pts_num * (3) + 
                                 boxes_idx * sampled_pts_num * (3) + 
                                 duplicate_idx * (3);
                int dst_offset = i * boxes_num * sampled_pts_num * (3) + 
                                 boxes_idx * sampled_pts_num * (3) + 
                                 k * (3);
                for (int j = 0; j < 3; j++)
                    pooled_features[dst_offset + j] = pooled_features[src_offset + j];
            }
        }
    }
}


__global__ void assign_pts_to_box3d(int batch_size, int pts_num, int boxes_num, const float *xyz, const float *boxes3d, int *pts_assign){
    // params xyz: (B, N, 3)
    // params boxes3d: (B, M, 7)
    // params pts_assign: (B, N, M): idx of the corresponding box3d, -1 means background points
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    int bs_idx = blockIdx.z;
    
    if (pt_idx >= pts_num || box_idx >= boxes_num || bs_idx >= batch_size){
        return;
    }
    int assign_idx = bs_idx * pts_num * boxes_num + pt_idx * boxes_num + box_idx;
    pts_assign[assign_idx] = 0;

    int box_offset = bs_idx * boxes_num * 7 + box_idx * 7;
    int pt_offset = bs_idx * pts_num * 3 + pt_idx * 3;
        
    int cur_in_flag = pt_in_box3d(xyz[pt_offset], xyz[pt_offset + 1], xyz[pt_offset + 2], boxes3d[box_offset], 
                                  boxes3d[box_offset + 1], boxes3d[box_offset + 2], boxes3d[box_offset + 3], 
                                  boxes3d[box_offset + 4], boxes3d[box_offset + 5], boxes3d[box_offset + 6], 10.0);

    pts_assign[assign_idx] = cur_in_flag;
    // printf("bs=%d, pt=%d, in=%d\n", bs_idx, pt_idx, pts_assign[bs_idx * pts_num + pt_idx]);
}


__global__ void get_pooled_idx(int batch_size, int pts_num, int boxes_num, int sampled_pts_num, 
                               const int *pts_assign, int *pts_idx, int *pooled_empty_flag){
    // params xyz: (B, N, 3)
    // params pts_assign: (B, N)
    // params pts_idx: (B, M, 512)
    // params pooled_empty_flag: (B, M)

    int boxes_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boxes_idx >= boxes_num){
        return;
    }

    int bs_idx = blockIdx.y;

    int cnt = 0;
    for (int k = 0; k < pts_num; k++){
        if (pts_assign[bs_idx * pts_num * boxes_num + k * boxes_num + boxes_idx]){
            if (cnt < sampled_pts_num){
                pts_idx[bs_idx * boxes_num * sampled_pts_num + boxes_idx * sampled_pts_num + cnt] = k;
                cnt++;
            }
            else break;
        }
    }

    if (cnt == 0){
        pooled_empty_flag[bs_idx * boxes_num + boxes_idx] = 1;
    }
    else if (cnt < sampled_pts_num){
        // duplicate same points for sampling
        for (int k = cnt; k < sampled_pts_num; k++){
            int duplicate_idx = k % cnt;
            int base_offset = bs_idx * boxes_num * sampled_pts_num + boxes_idx * sampled_pts_num;
            pts_idx[base_offset + k] = pts_idx[base_offset + duplicate_idx];
        }
    }
}


__global__ void roipool3d_forward(int batch_size, int pts_num, int boxes_num, int sampled_pts_num, 
                                   const float *xyz, const int *pts_idx, 
                                   float *pooled_features, int *pooled_empty_flag){
    // params xyz: (B, N, 3)
    // params pts_idx: (B, M, 512)
    // params pooled_features: (B, M, 512, 3+C)
    // params pooled_empty_flag: (B, M)
    
    int sample_pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    int bs_idx = blockIdx.z;
    
    if (sample_pt_idx >= sampled_pts_num || box_idx >= boxes_num || bs_idx >= batch_size){
        return;
    }

    if (pooled_empty_flag[bs_idx * boxes_num + box_idx]){
        return;
    }

    int temp_idx = bs_idx * boxes_num * sampled_pts_num + box_idx * sampled_pts_num + sample_pt_idx;
    int src_pt_idx = pts_idx[temp_idx];
    int dst_feature_offset = temp_idx * (3);

    for (int j = 0; j < 3; j++)
        pooled_features[dst_feature_offset + j] = xyz[bs_idx * pts_num * 3 + src_pt_idx * 3 + j];

}


void roipool3dLauncher_slow(int batch_size, int pts_num, int boxes_num, int sampled_pts_num, 
                       const float *xyz, const float *boxes3d, float *pooled_features, int *pooled_empty_flag){
    roipool3d_forward<<<DIVUP(boxes_num, THREADS_PER_BLOCK), 
                        THREADS_PER_BLOCK>>>(batch_size, pts_num, boxes_num, sampled_pts_num, 
                                             xyz, boxes3d, pooled_features, pooled_empty_flag);
    
#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}


void roipool3dLauncher(int batch_size, int pts_num, int boxes_num, int sampled_pts_num, 
                       const float *xyz, const float *boxes3d, float *pooled_features, int *pooled_empty_flag){

    // printf("batch_size=%d, pts_num=%d, boxes_num=%d\n", batch_size, pts_num, boxes_num);
    int *pts_assign = NULL;
    hipMalloc(&pts_assign, batch_size * pts_num * boxes_num * sizeof(int));  // (batch_size, N, M)
    // cudaMemset(&pts_assign, -1, batch_size * pts_num * boxes_num * sizeof(int));

    dim3 blocks(DIVUP(pts_num, THREADS_PER_BLOCK), boxes_num, batch_size);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);
    assign_pts_to_box3d<<<blocks, threads>>>(batch_size, pts_num, boxes_num, xyz, boxes3d, pts_assign);

    int *pts_idx = NULL;
    hipMalloc(&pts_idx, batch_size * boxes_num * sampled_pts_num * sizeof(int));  // (batch_size, M, sampled_pts_num)

    dim3 blocks2(DIVUP(boxes_num, THREADS_PER_BLOCK), batch_size);  // blockIdx.x(col), blockIdx.y(row)
    get_pooled_idx<<<blocks2, threads>>>(batch_size, pts_num, boxes_num, sampled_pts_num, pts_assign, pts_idx, pooled_empty_flag);

    dim3 blocks_pool(DIVUP(sampled_pts_num, THREADS_PER_BLOCK), boxes_num, batch_size); 
    roipool3d_forward<<<blocks_pool, threads>>>(batch_size, pts_num, boxes_num, sampled_pts_num, 
                                                      xyz, pts_idx, pooled_features, pooled_empty_flag);
    
    hipFree(pts_assign);
    hipFree(pts_idx);

#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}